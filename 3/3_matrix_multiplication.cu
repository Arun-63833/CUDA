#include <iostream>
#include <random>
#include <hip/hip_runtime.h>


using namespace std;

// CUDA kernel for matrix multiplication
__global__ void matmul(float *da, float *db, float *dc, int size) {
    int i = blockDim.y * blockIdx.y + threadIdx.y;
    int j = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < size && j < size) {
        float sum = 0;
        for (int k = 0; k < size; k++) {
            sum += da[i * size + k] * db[k * size + j];
        }
        dc[i * size + j] = sum;
    }
}

int main() {
    // Declare matrices
    float *a, *b, *c;
    int size = 16;
    int matrixSize = size * size;

    // Random number generation
    random_device rd;
    mt19937 gen(rd());
    uniform_real_distribution<> float_dist(0.0, 1.0);

    // Host memory allocation
    a = new float[matrixSize];
    b = new float[matrixSize];
    c = new float[matrixSize];

    // Initialize matrices a and b with random values
    for (int i = 0; i < matrixSize; i++) {
        a[i] = float_dist(gen);
        b[i] = float_dist(gen);
    }

    // Device memory allocation
    float *da, *db, *dc;
    hipError_t err;

    err = hipMalloc((void **)&da, matrixSize * sizeof(float));
    if (err != hipSuccess) {
        cerr << "Error while allocating memory for da: " << hipGetErrorString(err) << endl;
        exit(EXIT_FAILURE);
    }

    err = hipMalloc((void **)&db, matrixSize * sizeof(float));
    if (err != hipSuccess) {
        cerr << "Error while allocating memory for db: " << hipGetErrorString(err) << endl;
        exit(EXIT_FAILURE);
    }

    err = hipMalloc((void **)&dc, matrixSize * sizeof(float));
    if (err != hipSuccess) {
        cerr << "Error while allocating memory for dc: " << hipGetErrorString(err) << endl;
        exit(EXIT_FAILURE);
    }

    // Copy data from host to device
    err = hipMemcpy(da, a, matrixSize * sizeof(float), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        cerr << "Error while copying memory to da: " << hipGetErrorString(err) << endl;
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(db, b, matrixSize * sizeof(float), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        cerr << "Error while copying memory to db: " << hipGetErrorString(err) << endl;
        exit(EXIT_FAILURE);
    }

    // Define grid and block sizes
    dim3 block(8, 8, 1);
    dim3 grid((size + block.x - 1) / block.x, (size + block.y - 1) / block.y, 1);

    // Launch matrix multiplication kernel
    matmul<<<grid, block>>>(da, db, dc, size);

    // Copy result back to host
    err = hipMemcpy(c, dc, matrixSize * sizeof(float), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        cerr << "Error while copying memory to c: " << hipGetErrorString(err) << endl;
        exit(EXIT_FAILURE);
    }

    // Print the result matrix c
    for (int i = 0; i < matrixSize; i++) {
        if (i % size == 0) cout << endl;
        cout << c[i] << " ";
    }
    cout << endl;

    // Free device and host memory
    hipFree(da);
    hipFree(db);
    hipFree(dc);
    delete[] a;
    delete[] b;
    delete[] c;

    return 0;
}
